#include "hip/hip_runtime.h"
#include "ProjHelperFun.h"
#include "Constants.h"
#include "TridagPar.h"
#include "updateParamsKernels.cu.h"

#define BLOCK_DIM 8
void GPUimplicitX(PrivGlobs& globs, REAL* alist, REAL* blist, REAL* clist, const unsigned g){

  unsigned int block_dim = BLOCK_DIM;
  unsigned int numZ = max(globs.numX, globs.numY);
  REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);

  dim3 threadsPerBlock(block_dim, block_dim, 1);
  dim3 num_blocks(ceil((float)globs.numX/block_dim), ceil((float)globs.numY/block_dim));
  REAL* d_myVarX, *d_myDxx, *d_alist, *d_blist, *d_clist;

  hipMalloc((void**)&d_myVarX, globs.numX*globs.numY*sizeof(REAL));
  hipMalloc((void**)&d_myDxx, globs.numX*4*sizeof(REAL));
  hipMalloc((void**)&d_alist, numZ*numZ*sizeof(REAL));
  hipMalloc((void**)&d_blist, numZ*numZ*sizeof(REAL));
  hipMalloc((void**)&d_clist, numZ*numZ*sizeof(REAL));

  hipMemcpy(d_myVarX, globs.myVarX, globs.numX*globs.numY*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_myDxx, globs.myDxx, globs.numX*4*sizeof(REAL), hipMemcpyHostToDevice);

  implicitX<<<num_blocks, threadsPerBlock>>>(globs.numX, globs.numY, dtInv, d_myVarX, d_myDxx,
                                             d_alist, d_blist, d_clist);
  hipMemcpy(alist, d_alist, numZ*numZ*sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(blist, d_blist, numZ*numZ*sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(clist, d_clist, numZ*numZ*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_myVarX);
  hipFree(d_myDxx);
  hipFree(d_alist);
  hipFree(d_blist);
  hipFree(d_clist);

}

void GPUimplicitY(PrivGlobs& globs, REAL* alist, REAL* blist, REAL* clist, const unsigned g){

  unsigned int block_dim = BLOCK_DIM;
  unsigned int numZ = max(globs.numX, globs.numY);
  REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);

  dim3 threadsPerBlock(block_dim, block_dim, 1);
  dim3 num_blocks(ceil((float)globs.numX/block_dim), ceil((float)globs.numY/block_dim));
  REAL* d_myVarY, *d_myDyy, *d_alist, *d_blist, *d_clist;

  hipMalloc((void**)&d_myVarY, globs.numX*globs.numY*sizeof(REAL));
  hipMalloc((void**)&d_myDyy, globs.numY*4*sizeof(REAL));
  hipMalloc((void**)&d_alist, numZ*numZ*sizeof(REAL));
  hipMalloc((void**)&d_blist, numZ*numZ*sizeof(REAL));
  hipMalloc((void**)&d_clist, numZ*numZ*sizeof(REAL));

  hipMemcpy(d_myVarY, globs.myVarY, globs.numX*globs.numY*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_myDyy, globs.myDyy, globs.numY*4*sizeof(REAL), hipMemcpyHostToDevice);

  implicitY<<<num_blocks, threadsPerBlock>>>(globs.numX, globs.numY, dtInv, d_myVarY, d_myDyy,
                                             d_alist, d_blist, d_clist);
  hipMemcpy(alist, d_alist, numZ*numZ*sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(blist, d_blist, numZ*numZ*sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(clist, d_clist, numZ*numZ*sizeof(REAL), hipMemcpyDeviceToHost);

  hipFree(d_myVarY);
  hipFree(d_myDyy);
  hipFree(d_alist);
  hipFree(d_blist);
  hipFree(d_clist);

}

void updateParams(const unsigned g, const REAL alpha, const REAL beta, const REAL nu, PrivGlobs& globs)
{
        for(unsigned i=0;i<globs.numX;++i)
            for(unsigned j=0;j<globs.numY;++j) {
                globs.myVarX[i * globs.numY + j] =
                                              exp(2.0*(  beta*log(globs.myX[i])
                                            + globs.myY[j]
                                            - 0.5*nu*nu*globs.myTimeline[g] )
                                        );
                globs.myVarY[i * globs.numY + j] =
                                              exp(2.0*(  alpha*log(globs.myX[i])
                                            + globs.myY[j]
                                            - 0.5*nu*nu*globs.myTimeline[g] )
                                        ); // nu*nu
           }

}

void setPayoff(PrivGlobs& globs )
{
    //REAL* payoff = (REAL*) malloc(globs.outer * globs.numX*sizeof(REAL));
    for(unsigned h=0;h<globs.outer;h++)
	    for(unsigned i=0;i<globs.numX;++i) {
            for(unsigned j=0;j<globs.numY;++j)
		        globs.myResult[h * globs.numX * globs.numY + i * globs.numY + j] =
		                                          max(globs.myX[i]-0.001*h, (REAL)0.0);
            }


}

inline void tridag(
    REAL*   a,   // size [n]
    REAL*   b,   // size [n]
    REAL*   c,   // size [n]
    REAL*   r,   // size [n]
    const int             n,
          REAL*   u,   // size [n]
          REAL*   uu   // size [n] temporary
) {
    int    i;//, offset;
    REAL   beta;

    u[0]  = r[0];
    uu[0] = b[0];

    for(i=1; i<n; i++) {
        beta  = a[i] / uu[i-1];

        uu[i] = b[i] - beta*c[i-1];
        u[i]  = r[i] - beta*u[i-1];
    }

#if 1
    // X) this is a backward recurrence
    u[n-1] = u[n-1] / uu[n-1];
    for(i=n-2; i>=0; i--) {
        u[i] = (u[i] - c[i]*u[i+1]) / uu[i];
    }
#else
    // Hint: X) can be written smth like (once you make a non-constant)
    for(i=0; i<n; i++) a[i] =  u[n-1-i];
    a[0] = a[0] / uu[n-1];
    for(i=1; i<n; i++) a[i] = (a[i] - c[n-1-i]*a[i-1]) / uu[n-1-i];
    for(i=0; i<n; i++) u[i] = a[n-1-i];
#endif
 }


void
rollback( const unsigned g, const unsigned h, PrivGlobs& globs) {
    unsigned numX = globs.numX,
             numY = globs.numY;

    unsigned numZ = max(numX,numY);
    unsigned i, j;

    REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);
/*
    vector< vector<REAL> > u(numY, vector<REAL>(numX));   // [numY][numX]
    vector< vector<REAL> > v(numX, vector<REAL>(numY));   // [numX][numY]
    vector< vector<REAL> > alist(numZ, vector<REAL>(numZ));
    vector< vector<REAL> > blist(numZ, vector<REAL>(numZ));
    vector< vector<REAL> > clist(numZ, vector<REAL>(numZ));
    vector< vector<REAL> > ylist(numZ, vector<REAL>(numZ));
    vector<REAL> yy(numZ);  // temporary used in tridag  // [max(numX,numY)]
*/
    REAL* u = (REAL*) malloc(numY*numX*sizeof(REAL));
    REAL* v = (REAL*) malloc(numY*numX*sizeof(REAL));
    REAL* alist = (REAL*) malloc(numZ*numZ*sizeof(REAL));
    REAL* blist = (REAL*) malloc(numZ*numZ*sizeof(REAL));
    REAL* clist = (REAL*) malloc(numZ*numZ*sizeof(REAL));
    REAL* ylist = (REAL*) malloc(numZ*numZ*sizeof(REAL));
    REAL* yy = (REAL*) malloc(numZ*sizeof(REAL));

    unsigned int expand = numX * numY;

    //	explicit x

        for(i=0;i<numX;i++) {
            for(j=0;j<numY;j++) {
                u[j * numX + i] = dtInv*globs.myResult[h * expand + i * numY + j];

                if(i > 0) {
                  u[j * numX + i] += 0.5*( 0.5*globs.myVarX[i * numY + j]
                                           *globs.myDxx[i * 4 + 0])
                                           * globs.myResult[h * expand + (i-1) * numY + j];
                }
                u[j * numX + i]  +=  0.5*( 0.5*globs.myVarX[i * numY + j]
                                           *globs.myDxx[i * 4 + 1])
                                           *globs.myResult[h * expand + i * numY + j];
                if(i < numX-1) {
                  u[j * numX + i] += 0.5*( 0.5*globs.myVarX[i * numY + j]
                                           *globs.myDxx[i * 4 + 2])
                                           * globs.myResult[h * expand + (i+1) * numY + j];
                }
            }
        }


    //	explicit y
        for(j=0;j<numY;j++)
        {
            for(i=0;i<numX;i++) {
                v[ i* numY + j] = 0.0;

                if(j > 0) {
                  v[ i* numY + j] +=  ( 0.5*globs.myVarY[i * numY+ j]
                                               *globs.myDyy[j * 4 + 0] )
                                               *globs.myResult[h * expand + i * numY + (j-1)];
                }
                v[ i* numY + j]  +=   ( 0.5*globs.myVarY[i * numY +j]
                                               *globs.myDyy[j * 4 + 1] )
                                               *globs.myResult[h * expand + i * numY + j];
                if(j < numY-1) {
                  v[ i* numY + j] +=  ( 0.5*globs.myVarY[i * numY +j]
                                               *globs.myDyy[j * 4 + 2] )
                                               *globs.myResult[h * expand + i * numY + (j+1)];
                }
                u[ + j * numX + i] += v[ i * numY + j];
            }
        }

/*
    //	implicit x
        for(j=0;j<numY;j++) {
            for(i=0;i<numX;i++) {  // here a,b,c should have size [numX]
                alist[ j * numX + i] = - 0.5*(0.5*
                     globs.myVarX[ i * numY + j]*globs.myDxx[i * 4 + 0]);
                blist[ j * numX + i] = dtInv - 0.5*(0.5*
                     globs.myVarX[i * numY + j]*globs.myDxx[i * 4 + 1]);
                clist[ j * numX + i] = - 0.5*(0.5*
                     globs.myVarX[i * numY + j]*globs.myDxx[i * 4 + 2]);
            }
        //tridagPar(alist[j],blist[j],clist[j],u[j],numX,u[j],yy);
        // here yy should have size [numX]
        }
*/
        GPUimplicitX(globs, alist, blist, clist, g);
        for(j=0;j<numY;j++) {
            tridag(&alist[ j*numX],&blist[ j*numX],
                   &clist[ j*numX],&u[ j*numX],
                    numX,&u[j*numX],yy);
        }

    //	implicit y
        for(i=0;i<numX;i++) {
            for(j=0;j<numY;j++) {  // here a, b, c should have size [numY]
                alist[ i * numY + j] = - 0.5*(0.5*globs.myVarY[i * numY + j]
                                                  *globs.myDyy[j * 4 + 0]);
                blist[ i * numY + j] = dtInv - 0.5*(0.5
                                                  *globs.myVarY[i * numY + j]
                                                  *globs.myDyy[j * 4 + 1]);
                clist[ i * numY + j] = - 0.5*(0.5
                                                  *globs.myVarY[i * numY + j]
                                                  *globs.myDyy[j * 4 + 2]);
            }
        }


        // here yy should have size [numY]
      //  GPUimplicitY(globs, alist, blist, clist, g);
        for(i=0;i<numX;i++) {
            for(j=0;j<numY;j++) {
                ylist[ i * numY + j] = dtInv*u[ + j * numX + i]
                                             - 0.5*v[ i * numY + j];
            }
        }

        for(i=0;i<numX;i++) {
            tridag(&alist[ i*numY],&blist[ i*numY],
                   &clist[ i*numY],&ylist[ i*numY],
                    numY,&globs.myResult[h * expand + i*numY],yy);
        }

}

REAL   value(   PrivGlobs    globs,
                const REAL s0,
                const REAL strike,
                const REAL t,
                const REAL alpha,
                const REAL nu,
                const REAL beta,
                const unsigned int numX,
                const unsigned int numY,
                const unsigned int numT
) {
    initGrid(s0,alpha,nu,t, numX, numY, numT, globs);
    initOperator(globs.myX,globs.myDxx, numX);
    initOperator(globs.myY,globs.myDyy, numY);

    setPayoff(globs);
    for(int i = numT-2;i>=0;--i)
    {
        updateParams(i,alpha,beta,nu,globs);
        //rollback(i, 2, globs);
    }

    return globs.myResult[globs.myXindex * numY + globs.myYindex];
}

void GPUupdateParams(const unsigned g, const REAL alpha, const REAL beta,
                                         const REAL nu, PrivGlobs& globs)
{

    unsigned int block_dim = BLOCK_DIM;
    dim3 threadsPerBlock(block_dim, block_dim, 1);
    dim3 num_blocks(ceil((float)globs.numX/block_dim), ceil((float)globs.numY/block_dim),1);

    REAL *d_myVarX, *d_myVarY, *d_myX, *d_myY, *d_myTimeline;
    hipMalloc((void**)&d_myVarX, globs.numX*globs.numY*sizeof(REAL));
    hipMalloc((void**)&d_myVarY, globs.numX*globs.numY*sizeof(REAL));
    hipMalloc((void**)&d_myX, globs.numX*sizeof(REAL));
    hipMalloc((void**)&d_myY, globs.numY*sizeof(REAL));
    hipMalloc((void**)&d_myTimeline, globs.numT*sizeof(REAL));

    hipMemcpy(d_myVarX, globs.myVarX, globs.numX*globs.numY*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myVarY, globs.myVarY, globs.numX*globs.numY*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myX, globs.myX, globs.numX*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myY, globs.myY, globs.numY*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myTimeline, globs.myTimeline,
               globs.numT*sizeof(REAL), hipMemcpyHostToDevice);

    updateParamsKer<<<num_blocks, threadsPerBlock>>>(g, alpha, beta, nu, globs.numX, globs.numY,
                                   d_myX, d_myY, d_myVarX, d_myVarY, d_myTimeline);

    hipDeviceSynchronize();

    hipMemcpy(globs.myVarX, d_myVarX, globs.numX*globs.numY*sizeof(REAL), hipMemcpyDeviceToHost);
    hipMemcpy(globs.myVarY, d_myVarY, globs.numX*globs.numY*sizeof(REAL), hipMemcpyDeviceToHost);


    hipFree(d_myVarX);
    hipFree(d_myVarY);
    hipFree(d_myX);
    hipFree(d_myY);
    hipFree(d_myTimeline);


}

void GPUsetParams(PrivGlobs& globs)
{

    unsigned int block_dim = BLOCK_DIM;
    dim3 threadsPerBlock(block_dim, block_dim, 1);
    dim3 num_blocks(ceil((float)globs.numX/block_dim), ceil((float)globs.numY/block_dim),globs.outer);

    REAL *d_myX, *d_myResult;
    hipMalloc((void**)&d_myResult, globs.outer*globs.numX*globs.numY*sizeof(REAL));
    hipMalloc((void**)&d_myX, globs.numX*sizeof(REAL));

    hipMemcpy(d_myResult, globs.myResult, globs.outer*globs.numX*globs.numY*sizeof(REAL), hipMemcpyHostToDevice);
    hipMemcpy(d_myX, globs.myX, globs.numX*sizeof(REAL), hipMemcpyHostToDevice);

    setParamsKer<<<num_blocks, threadsPerBlock>>>(globs.numX, globs.numY, d_myX, d_myResult);

    hipDeviceSynchronize();

    hipMemcpy(globs.myResult, d_myResult, globs.outer*globs.numX*globs.numY*sizeof(REAL), hipMemcpyDeviceToHost);

    hipFree(d_myX);
    hipFree(d_myResult);


}

void   run_OrigCPU(
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t,
                const REAL&           alpha,
                const REAL&           nu,
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {

    PrivGlobs globs(numX, numY, numT, outer);
/*
    for(unsigned i = 0; i < outer;++i) {
        strike[i] = 0.001*i;
        //PrivGlobs globs(numX, numY, numT, outer);
        //globslist[i] = globs;

    }*/

    initGrid(s0,alpha,nu,t, numX, numY, numT, globs);
    initOperator(globs.myX,globs.myDxx, numX);
    initOperator(globs.myY,globs.myDyy, numY);

    //setPayoff(globs);
    GPUsetParams(globs);
    for(int h=0; h<outer; h++) {
        for(int i = numT-2;i>=0;--i)
        {
            GPUupdateParams(i,alpha,beta,nu,globs);
            rollback(i, h, globs);
        }
    }

    for(unsigned i = 0; i < outer; ++i){
        res[i] = globs.myResult[i * numX * numY + globs.myXindex * numY + globs.myYindex];
    }
/*
    for( unsigned i = 0; i < outer; ++ i ) {
        res[i] = value( globslist[i], s0, strike[i], t,
                        alpha, nu,    beta,
                        numX,  numY,  numT );
    }*/
}

//#endif // PROJ_CORE_ORIG
